#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

__device__ float4 juliabulb(const float3 dim_limits, const float r, const float theta, const float phi)
{
    float4 out_coords;
    out_coords.s3 = r * r * r * r * r * r * r * r;
    out_coords.s0 = dim_limits.s0 + out_coords.s3 * cos(theta) * cos(phi) + 0.353;
    out_coords.s1 = dim_limits.s1 + out_coords.s3 * sin(theta) * cos(phi) + 0.288;
    out_coords.s2 = dim_limits.s2 + out_coords.s3 * sin(phi) + 0.2;
    return out_coords;
}

__device__ float4 mandelbulb(const float3 dim_limits, const float r, const float theta, const float phi)
{
    float4 out_coords;
    out_coords.s3 = r * r * r * r * r * r * r * r;
    out_coords.s0 = dim_limits.s0 + out_coords.s3 * cos(theta) * cos(phi);
    out_coords.s1 = dim_limits.s1 + out_coords.s3 * sin(theta) * cos(phi);
    out_coords.s2 = dim_limits.s2 + out_coords.s3 * sin(phi);
    return out_coords;
}

template <int FRACTAL_ID> 
__global__ void fractal3d
         (int* restrict image,
          const int depth_idx,
          const int3 dimensions,
          const int2 INT_CONSTANTS,
          const float3 FLT_CONSTANTS)
{
    const float MIN_LIMIT = FLT_CONSTANTS.s0;
    const float MAX_LIMIT = FLT_CONSTANTS.s1;
    const int ORDER = INT_CONSTANTS.s1;

    float3 dim_limits;
    dim_limits.s0 = MIN_LIMIT + get_global_id(0) * ((MAX_LIMIT - MIN_LIMIT) / dimensions.s0);
    dim_limits.s1 = MIN_LIMIT + get_global_id(1) * ((MAX_LIMIT - MIN_LIMIT) / dimensions.s1);
    dim_limits.s2 = MIN_LIMIT + depth_idx * ((MAX_LIMIT - MIN_LIMIT) / dimensions.s2);

    float4 coords;
    coords.s0 = 0.0f;
    coords.s1 = 0.0f;
    coords.s2 = 0.0f;
    coords.s3 = 0.0f;

    float r = 0.0f;
    float theta = 0.0f;
    float phi = 0.0f;
    int iter_num = 0;
    int i = 0;
    for (iter_num = 0; iter_num < INT_CONSTANTS.s0; ++iter_num)
    {
        r = sqrt(coords.s0 * coords.s0 + coords.s1 * coords.s1 + coords.s2 * coords.s2);
        if(r > FLT_CONSTANTS.s2)
            break;

        theta = ORDER * atan2(sqrt(coords.s0 * coords.s0 + coords.s1 * coords.s1), coords.s2);
        phi =   ORDER * atan2(coords.s0, coords.s1);
        
        //use the given fractal type
        switch(FRACTAL_ID)
        {
          case MANDELBROT:
          {
  				  coords = mandelbulb(dim_limits, r, theta, phi);  
            break;
          }
          case JULIA:
          {
  				  coords = juliabulb(dim_limits, r, theta, phi);  
            break;
          }        
        }
    }
    
    image[threadIdx.x + blockIdx.x*blockDim.x] = max(0, iter_num-1);
}                      




