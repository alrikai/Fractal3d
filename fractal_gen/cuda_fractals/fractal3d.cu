#include "hip/hip_runtime.h"
#include "fractal3d.h"

/* fractal3d.cu -- part of the CUDA fractal3d implementation 
 *
 * Copyright (C) 2015 Alrik Firl 
 *
 * This software may be modified and distributed under the terms
 * of the MIT license.  See the LICENSE file for details.
 */

__device__ float4 juliabulb(const float3 dim_limits, const float r, const float theta, const float phi)
{
    float4 out_coords;
    out_coords.w = r * r * r * r * r * r * r * r;
    out_coords.x = dim_limits.x + out_coords.w * cos(theta) * cos(phi) + 0.353;
    out_coords.y = dim_limits.y + out_coords.w * sin(theta) * cos(phi) + 0.288;
    out_coords.z = dim_limits.z + out_coords.w * sin(phi) + 0.2;
    return out_coords;
}

__device__ float4 mandelbulb(const float3 dim_limits, const float r, const float theta, const float phi)
{
    float4 out_coords;
    out_coords.w = r * r * r * r * r * r * r * r;
    out_coords.x = dim_limits.x + out_coords.w * cos(theta) * cos(phi);
    out_coords.y = dim_limits.y + out_coords.w * sin(theta) * cos(phi);
    out_coords.z = dim_limits.z + out_coords.w * sin(phi);
    return out_coords;
}

//TODO: implement some other fractal functions here....

template <typename pixel_t, int FRACTAL_ID> 
__global__ void fractal3d_kernel (pixel_t* image,
                           const int depth_idx,
                           const int4 dimensions,
                           const int2 INT_CONSTANTS,
                           const float4 FLT_CONSTANTS)
{
    const float MIN_LIMIT = FLT_CONSTANTS.x;
    const float MAX_LIMIT = FLT_CONSTANTS.y;
    const int ORDER = INT_CONSTANTS.y;

		const int index_x = blockIdx.x * blockDim.x + threadIdx.x;    
    const int index_y = blockIdx.y * blockDim.y + threadIdx.y;

    float3 dim_limits;
    dim_limits.x = MIN_LIMIT + index_x * ((MAX_LIMIT - MIN_LIMIT) / dimensions.x);
    dim_limits.y = MIN_LIMIT + index_y * ((MAX_LIMIT - MIN_LIMIT) / dimensions.y);
    dim_limits.z = MIN_LIMIT + depth_idx * ((MAX_LIMIT - MIN_LIMIT) / dimensions.z);

    float4 coords = (float4) {0.0f, 0.0f, 0.0f, 0.0f};

    float r = 0.0f;
    float theta = 0.0f;
    float phi = 0.0f;
    pixel_t iter_num = 0;
    for (iter_num = 0; iter_num < INT_CONSTANTS.x; ++iter_num)
    {
        r = sqrt(coords.x * coords.x + coords.y * coords.y + coords.z * coords.z);
        if(r > FLT_CONSTANTS.z)
            break;

        theta = ORDER * atan2(sqrt(coords.x * coords.x + coords.y * coords.y), coords.z);
        phi   = ORDER * atan2(coords.x, coords.y);
        
        //use the given fractal type
        switch(FRACTAL_ID)
        {
          case MANDELBROT:
          {
  				  coords = mandelbulb(dim_limits, r, theta, phi);  
            break;
          }
          case JULIA:
          {
  				  coords = juliabulb(dim_limits, r, theta, phi);  
            break;
          }        
        }
    }
    image[index_y * dimensions.x + index_x] = max(0, iter_num-1);
}

//--------------------------------------------------------------------------------------------------------------------------------

template <typename pixel_t, int FRACTAL_ID> __host__ 
void run_fractalgen(pixel_t* dev_image, int depth_idx, const int4 dimensions, const int2 constants, const float4 flt_constants)
{
  static constexpr int blockdim = 16;
  //want to process a frame per kernel invocation -- frames will be something e.g. [128 x 128], [512 x 512], [1024 x 1024], etc. 
  dim3 block_dim (blockdim, blockdim);
  dim3 grid_dim  (static_cast<int>(std::ceil(dimensions.x / static_cast<float>(blockdim))), static_cast<int>(std::ceil(dimensions.y / static_cast<float>(blockdim))));

  fractal3d_kernel<pixel_t, FRACTAL_ID><<<grid_dim, block_dim>>> (dev_image, depth_idx, dimensions, constants, flt_constants);
}

template __host__ void run_fractalgen <unsigned char, 0> (unsigned char*, int, const int4, const int2, const float4);
template __host__ void run_fractalgen <unsigned char, 1> (unsigned char*, int, const int4, const int2, const float4);

